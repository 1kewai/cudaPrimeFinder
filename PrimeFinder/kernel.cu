#include "hip/hip_runtime.h"
﻿//CUDA Cのinclude
#include "hip/hip_runtime.h"
#include ""

//cpu側include
#include <stdio.h>
#include <stdlib.h>

//CUDA How-to memo
//スレッドの固有の番号を取得:blockIdx.x * blockDim.x + threadIdx.x
//メモリの内容をコピー:hipMemcpy(HostMemory, Device_array, num_of_rnd * 4, hipMemcpyDeviceToHost);
//CUDAの処理完了を待機:hipDeviceSynchronize();
//デバイスメモリ確保:int* Device_addr; hipMalloc((void**)&Device_addr, サイズ);
//デバイスメモリ解放;hipFree(ポインタ)
//メモリコピー:hipMemcpy(HostMemory(dst), Device_array(src), num_of_rnd * 4, hipMemcpyDeviceToHost);

//素数を保存するためのテーブルのサイズ
const int table_size = 1024;
//素数テーブルの数
const int table_n = 512;
//結果をGPUとやり取りするためのフラグ変数のアドレス
short* flag;
short t = short(1);
short f = short(0);

//GPUコード
//渡された数値が指定されたアドレスの素数テーブルにある数値で割り切れるか確かめる。もし割り切れるなら結果フラグを1, 割り切れないなら0にする。
__global__ void Device_PrimeChk(long number, long* table, short* flag ){
	if (number % table[blockIdx.x * blockDim.x + threadIdx.x] == 0) {
		*flag = short(1);
	}
}

//渡されたアドレスから、渡された数だけlongの数を読み取ってprintf
__global__ void debug(long* table, int number) {
	for (int i = 0; i < number; i++) {
		printf("%ld\n", table[i]);
	}
}

//CPUコード
//テーブルを初期化する
//最初の中身は素数である2で初期化を行う
void Host_init_table(long* table) {
	for (int i = 0; i < table_size; i++) {
		table[i] = long(2);
	}
}

int main(){
	//GPUの素数テーブルのアドレステーブル部分をcpu側スタックに確保
	long* Device_table[table_n];

	//素数テーブルの雛形をcpuで作成、中身を2で埋める
	long* table_template;
	table_template = new long[table_size];
	Host_init_table(table_template);

	//GPU側にテーブルのテンプレートを必要数だけコピー
	for (int i = 0; i < table_n; i++) {
		hipMalloc((void**)&Device_table[i], 8 * table_size);
		hipMemcpy(Device_table[i], table_template, 8 * table_size, hipMemcpyHostToDevice);
	}

	//GPU側に結果フラグを書き込んでいくメモリ領域を作成し０を書き込む
	hipMalloc((void**)&flag, 2);
	hipMemcpy(flag, (void**)&f, 2, hipMemcpyHostToDevice);
	//CPU側からはテーブルのテンプレートを削除
	free(table_template);

	//実際に計算を行う。
	//GPU側でそれまで発見された素数を使って割り算=>もし成功したらcpuがそれをポーリング=>GPUメモリに書き込むことで素数リストを更新
	int page = 0;//素数を次に記録するページ
	int index = 1;//素数を次に記録する位置
	long search = 3;//次に素数かどうかを検証する位置
	
	int i = 0;
	bool chk = false;
	while (1) {
		chk = false;
		//ページ数が複数ある場合、すべてのページに対して割り切れる数が無いか探索を行う
		short Host_flag = f;
		for (i = 0; i <= page; i++) {
			//GPUに演算を投げる
			Device_PrimeChk << <1, table_size >> > (search, Device_table[i], flag);
			hipDeviceSynchronize();
			hipMemcpy(&Host_flag, flag, 2, hipMemcpyDeviceToHost);
			if (Host_flag != f) {
				search++;
				chk = true;
				hipMemcpy(flag, (void**)&f, 2, hipMemcpyHostToDevice);
				continue;
			}
		}
		if (chk == false) {
			if (index == table_size) {
				index = 0;
				page++;
			}
			printf("%ld\n", search);
			hipMemcpy(&Device_table[page][index], &search, 8, hipMemcpyHostToDevice);
			index++;
			search++;
		}
	}
}